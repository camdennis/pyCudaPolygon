#include <hip/hip_runtime.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <complex>
#include <hiprand/hiprand_kernel.h>
#include <float.h>

static const dim3 myBlockDim(16, 16);
static const int blockSize = 256;


__global__ void initStatesKernel(hiprandState *globalState, unsigned long long seed, int gridSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Flatten 2D index to 1D index
    int idx = row * gridSize + col;
    if (row < gridSize && col < gridSize) {
       hiprand_init(seed, idx, 0, &globalState[idx]);
    }
}

extern "C" void initializeRandomStates(hiprandState *globalState, unsigned long long int seed, int gridSize) {
    dim3 gridDim((gridSize + myBlockDim.x - 1) / myBlockDim.x, (gridSize + myBlockDim.y - 1) / myBlockDim.y);
    initStatesKernel<<<gridDim, myBlockDim>>>(globalState, seed, gridSize);
    hipDeviceSynchronize();
}